#include "hip/hip_runtime.h"
#include <string>
#include <vector>
#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <cstring>

#include <sys/stat.h>
#include <boost/numeric/odeint.hpp>
#include <boost/numeric/odeint/external/openmp/openmp.hpp>
#include <boost/random.hpp>

typedef double prec;


#include "Ensemble_param.h"
#include "Ensemble_connections.h"
#include "Ensemble_Dinamic.h"
#include "Ensemble_evolve.h"

void Evolve::translate_init()
{
	for (int i = 0; i < N; ++i)
	{
		translated_init[2*i]=x_i[i];
		translated_init[2*i+1]=v_i[i];
		if(i==0)
		{
			int entero=translated_init[2*i]/(2.0*M_PI);
			translated_init[2*i]=translated_init[2*i]-2*M_PI*entero;
		}
	}
}

void Evolve::run(Dinamic &P, prec t_start, prec t_end, prec t_save)
{
	if(t_save<0)
	{
		t_save=t_end;
	}
	std::cout << std::endl;
	std::cout << "N=" << N << std::endl;
	std::cout << "Running..." << std::endl;


	boost::numeric::odeint::runge_kutta4 <std::vector< prec > , prec , std::vector< prec > , prec , boost::numeric::odeint::openmp_range_algebra> stepper;
	push_back_state_and_time push(q , t, t_start, t_end, t_save);
	size_t steps=integrate_adaptive(stepper, P(), translated_init , t_start , t_end, 0.01 , push);
	std::cout << std::endl;
}

int Evolve::find_next_maxima(int t_start,int k)
{
	while(v(k,t_start+1)<=v(k,t_start) && t_start < t.size()-2)
	{
		t_start++;
	}
	if(v(k,t_start+1)>v(k,t_start))
	{
		for (int i = t_start; i < t.size()-1; ++i)
		{
			if(v(k,i+1)<=v(k,i))
			{
				return i;
			}
		}
	}
	return t.size()-1;
}

int Evolve::find_next_minima(int t_start,int k)
{
	while(v(k,t_start+1)>=v(k,t_start) && t_start < t.size()-2)
	{
		t_start++;
	}
	if(v(k,t_start+1)<v(k,t_start))
	{
		for (int i = t_start; i < t.size()-1; ++i)
		{
			if(v(k,i+1)>=v(k,i))
			{
				return i;
			}
		}
	}
	return t.size()-1;
}

bool Evolve::did_converge(int i)
{
	if(convergence(i)>conv_crit)
	{
		return true;
	}
	else
	{
		return false;
	}
}

prec Evolve::convergence(int k)
{
	int first_maxima=find_next_maxima(0,k);
	if(first_maxima<t.size()-1)
	{
		int second_maxima=find_next_maxima(first_maxima+1,k);
		int first_minima=find_next_minima(first_maxima+1,k);
		if(second_maxima<t.size()-1 && first_minima<t.size()-1)
		{
			return 100-100*fabs((v(k,second_maxima)-v(k,first_maxima))/(v(k,first_maxima)-v(k,first_minima)));
		}
	}
	return 0;
}

prec Evolve::period(int k)
{
	if(convergence(k)>conv_crit)
	{
		int first_maxima=find_next_maxima(0,k);
		int second_maxima=find_next_maxima(first_maxima+1,k);
		return t[second_maxima]-t[first_maxima];
	}
	return 0;
}

prec Evolve::frec(int k)
{
	prec ret=period(k);
	if(ret>0)
	{
		return 2*M_PI/ret;
	}
	return 0;
}

prec Evolve::drift(int k)
{
	if(convergence(k)>conv_crit)
	{
		int first_maxima=find_next_maxima(0,k);
		if(first_maxima<t.size()-1)
		{
			int last_maxima=find_next_maxima(first_maxima+1,k);
			int moving=last_maxima;
			
			while(moving<t.size()-1)
			{
				last_maxima=moving;
				moving=find_next_maxima(moving+1,k);
			}
			if(first_maxima==last_maxima)
			{
				return 0;
			}
			return (x(k,last_maxima)-x(k,first_maxima))/(t[last_maxima]-t[first_maxima]);
		}
	}
	return 0;
}

void Evolve::print_init()
{
	x_i.print();
	v_i.print();
}

void Evolve::load_init()
{
	x_i.load();
	v_i.load();
	translate_init();
}

void Evolve::reset_init()
{
	for (int i = 0; i < N; ++i)
	{
		x_i.assign(i,x(i,q.size()-1));
		v_i.assign(i,v(i,q.size()-1));
	}
	translate_init();
}

prec Evolve::x(int i,int t_i)
{
	return q[t_i][2*i];
}

prec Evolve::v(int i,int t_i)
{
	return q[t_i][2*i+1];
}

void Evolve::print_x()
{
	mkdir("out", 0777);

	std::ofstream txtOut;
	txtOut.precision(std::numeric_limits< prec >::max_digits10);
	txtOut.open("out/X.txt");
	for (int j = 0; j < t.size(); ++j)
	{
		txtOut << t[j] << " ";
		for (int i = 0; i < N; ++i)
		{
			txtOut << x(i,j) << " ";
		}
		txtOut << std::endl;
	}
	txtOut.close();
}

void Evolve::print_v()
{
	mkdir("out", 0777);

	std::ofstream txtOut;
	txtOut.precision(std::numeric_limits< prec >::max_digits10);
	txtOut.open("out/V.txt");
	for (int j = 0; j < t.size(); ++j)
	{
		txtOut << t[j] << " ";
		for (int i = 0; i < N; ++i)
		{
			txtOut << v(i,j) << " ";
		}
		txtOut << std::endl;
	}
	txtOut.close();
}

void Evolve::print()
{
	std::cout << "Printing X..." << std::endl;
	print_x();
	std::cout << "Printing V..." << std::endl;
	print_v();
}

void Evolve::clean()
{
	q.clear();
	t.clear();

}

int Evolve::size()
{
	return N;
}