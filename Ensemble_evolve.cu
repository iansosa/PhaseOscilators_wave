#include "hip/hip_runtime.h"
#include <string>
#include <vector>
#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <cstring>

#include <sys/stat.h>
#include <boost/numeric/odeint.hpp>
#include <boost/numeric/odeint/external/openmp/openmp.hpp>
#include <boost/random.hpp>

typedef double prec;

#include "Ensemble_param.h"
#include "Ensemble_connections.h"
#include "Ensemble_Dinamic.h"
#include "Ensemble_evolve.h"

void Evolve::translate_init()
{
	for (int i = 0; i < N; ++i)
	{
		translated_init[2*i]=x_i[i];
		translated_init[2*i+1]=v_i[i];
		if(i==0)
		{
			int entero=translated_init[2*i]/(2.0*M_PI);
			translated_init[2*i]=translated_init[2*i]-2*M_PI*entero;
		}
	}
}

void Evolve::run(Dinamic &P, prec t_start, prec t_end, prec t_save)
{
	if(t_save<0)
	{
		t_save=t_end;
	}
	std::cout << std::endl;
	std::cout << "N=" << N << std::endl;
	std::cout << "Running..." << std::endl;

	prec dt=0.01;
	boost::numeric::odeint::runge_kutta4 <std::vector< prec > , prec , std::vector< prec > , prec , boost::numeric::odeint::openmp_range_algebra> stepper;
	push_back_state_and_time push(q , t, t_start, t_end, t_save,dt);
	
	size_t steps=integrate_adaptive(stepper, P(), translated_init , t_start , t_end, dt , push);
	std::cout << std::endl;
}

int Evolve::find_next_maxima(int t_start,int k)
{
	while(v(k,t_start+1)<=v(k,t_start) && t_start < t.size()-2)
	{
		t_start++;
	}
	if(v(k,t_start+1)>v(k,t_start))
	{
		for (int i = t_start; i < t.size()-1; ++i)
		{
			if(v(k,i+1)<=v(k,i))
			{
				return i;
			}
		}
	}
	return t.size()-1;
}

int Evolve::find_next_minima(int t_start,int k)
{
	while(v(k,t_start+1)>=v(k,t_start) && t_start < t.size()-2)
	{
		t_start++;
	}
	if(v(k,t_start+1)<v(k,t_start))
	{
		for (int i = t_start; i < t.size()-1; ++i)
		{
			if(v(k,i+1)>=v(k,i))
			{
				return i;
			}
		}
	}
	return t.size()-1;
}

bool Evolve::did_converge(int i)
{
	if(convergence(i)>conv_crit)
	{
		return true;
	}
	else
	{
		return false;
	}
}

prec Evolve::convergence(int k)
{
	int first_maxima=find_next_maxima(0,k);
	if(first_maxima<t.size()-1)
	{
		int second_maxima=find_next_maxima(first_maxima+1,k);
		int first_minima=find_next_minima(first_maxima+1,k);
		if(second_maxima<t.size()-1 && first_minima<t.size()-1 && second_maxima-first_minima>50)
		{
			prec first_v=v(k,first_maxima);
			prec second_v=v(k,first_minima);
			if(first_v>second_v)
			{
				return 100-100*fabs((v(k,second_maxima)-v(k,first_maxima))/(first_v-second_v));
			}
		}
	}
	return 0;
}

prec Evolve::period(int k)
{
	if(convergence(k)>conv_crit)
	{
		int first_maxima=find_next_maxima(0,k);
		int second_maxima=find_next_maxima(first_maxima+1,k);
		return t[second_maxima]-t[first_maxima];
	}
	return 0;
}

prec Evolve::frec(int k)
{
	prec ret=period(k);
	if(ret>0)
	{
		return 2*M_PI/ret;
	}
	return 0;
}

prec Evolve::drift(int k)
{
	//std::cout << k << " entered w" << std::endl;
	if(convergence(k)>conv_crit)
	{
		//std::cout << "drift: " << k << " passed conv test" << std::endl;
		int first_maxima=find_next_maxima(1,k);
		//std::cout <<"drift: K:" << k << " first_maxima: " << first_maxima;
		if(first_maxima<t.size()-1)
		{
			int last_maxima=find_next_maxima(first_maxima+1,k);
			
			int moving=last_maxima;
			
			while(moving<t.size()-1)
			{
				last_maxima=moving;
				moving=find_next_maxima(moving+1,k);
			}
			//std::cout << "last_maxima: " << last_maxima << std::endl;
			if(first_maxima==last_maxima)
			{
				return 0;
			}
			return (x(k,last_maxima)-x(k,first_maxima))/(t[last_maxima]-t[first_maxima]);
		}
	}
	return 0;
}

prec Evolve::Amp(int k,Dinamic &P)
{
	if(did_converge(k)==true)
	{
		//std::cout << k << " did converge" << std::endl;
		int tstart=find_next_maxima(0,k);
		//std::cout << "amp: first maxima " << tstart << " v " << v(k,tstart);
		int tend=find_next_maxima(tstart+1,k);
		//std::cout << " second maxima " << tend << " v" << v(k,tend) << std::endl;
		if(tstart==tend)
		{
			return 0;
		}
		prec maxima=-10000;
		prec minima=100000;
		prec current;
		prec w;
		if(P.get_type()=="h_chain")
		{
			w=0;
		}
		else
		{
			w=drift(k);
		}
		for (int i = tstart; i < tend; ++i)
		{
			
			current=x(k,i)-w*t[i];
			if(maxima<current)
			{
				maxima=current;
			}
			if(minima>current)
			{
				minima=current;
			}
		}
		return (maxima-minima)/2.0;
	}
	return 0;
}

prec Evolve::Diff(int k)
{
	if(did_converge(k)==true && k<N-1 && k>0)
	{
		int tstart=find_next_maxima(0,k);
		int tend=find_next_maxima(tstart+1,k);
		if(tstart==tend)
		{
			return 0;
		}
		prec maxima=-10000;
		prec current;
		for (int i = tstart; i < tend; ++i)
		{
			current=fabs(x(k,i)-x(k+1,i));
			if(maxima<current)
			{
				maxima=current;
			}
		}
		return maxima;
	}
	return 0;
}

void Evolve::print_init()
{
	x_i.print();
	v_i.print();
}

void Evolve::load_init()
{
	x_i.load();
	v_i.load();
	translate_init();
}

void Evolve::reset_init()
{
	for (int i = 0; i < N; ++i)
	{
		x_i.assign(i,x(i,q.size()-1));
		v_i.assign(i,v(i,q.size()-1));
	}
	translate_init();
}

prec Evolve::x(int i,int t_i)
{
	return q[t_i][2*i];
}

prec Evolve::v(int i,int t_i)
{
	return q[t_i][2*i+1];
}

void Evolve::print_x()
{
	mkdir("out", 0777);

	std::ofstream txtOut;
	txtOut.precision(std::numeric_limits< prec >::max_digits10);
	txtOut.open("out/X.txt");
	for (int j = 0; j < t.size(); ++j)
	{
		txtOut << t[j] << " ";
		for (int i = 0; i < N; ++i)
		{
			txtOut << x(i,j) << " ";
		}
		txtOut << std::endl;
	}
	txtOut.close();
}

void Evolve::print_v()
{
	mkdir("out", 0777);

	std::ofstream txtOut;
	txtOut.precision(std::numeric_limits< prec >::max_digits10);
	txtOut.open("out/V.txt");
	for (int j = 0; j < t.size(); ++j)
	{
		txtOut << t[j] << " ";
		for (int i = 0; i < N; ++i)
		{
			txtOut << v(i,j) << " ";
		}
		txtOut << std::endl;
	}
	txtOut.close();
}

void Evolve::print()
{
	std::cout << "Printing X..." << std::endl;
	print_x();
	std::cout << "Printing V..." << std::endl;
	print_v();
}

void Evolve::clean()
{
	q.clear();
	t.clear();

}

int Evolve::size()
{
	return N;
}