#include "Ensemble_param.h"


void Ensemble_param::dist(prec in_mid, prec in_sigma)
{
	mid=in_mid;
	sigma=in_sigma;
}

int Ensemble_param::size()
{
	return N;
}

void Ensemble_param::resize(int in_N)
{
	N=in_N;
	container.resize(in_N);
	generate(false);
}

void Ensemble_param::generate(bool first)
{
	boost::normal_distribution<> gauss(mid,sigma);
    boost::variate_generator< boost::mt19937&, boost::normal_distribution<> > gen(rng,gauss);
    for (int i = 0; i < container.size(); ++i)
    {
    	container[i]=gen();
    }

    if(first==false)
    {
    	container[0]=mid;
    }

    if(type=="ofirst")
    {
    	container[0]=mid;
    	for (int i = 1; i < container.size(); ++i)
    	{
    		container[i]=0;
    	}
    }
}

void Ensemble_param::print()
{
	mkdir("params", 0777);

	std::ofstream txtOut;
	txtOut.open("params/"+name+".txt");
	txtOut.precision(8);
	txtOut << N << " ";
	txtOut << mid << " ";
	txtOut << sigma << " ";
	txtOut << type << " ";
	for (int i = 0; i < container.size(); ++i)
	{
		txtOut << container[i] << " ";
	}
	txtOut.close();
}

void Ensemble_param::load()
{
	std::ifstream txtIn_check;
	int N_check;
	txtIn_check.open("params.txt");
	txtIn_check >> N_check;
	txtIn_check.close();	
	std::ifstream txtIn;
	txtIn.open("params/"+name+".txt");
	txtIn >> N;
	txtIn >> mid;
	txtIn >> sigma;
	txtIn >> type;
	container.resize(N);
	if(N_check!=N)
	{
		std::cout << "INCONSISTENT PARAM SIZE FOR " << name << ". Expects N=" << N << " but check throws N=" << N_check <<std::endl;
	}
	for (int i = 0; i < container.size(); ++i)
	{
		txtIn >> container[i];
	}
	txtIn.close();
}

void Ensemble_param::type_rand(bool is_rand)
{
	if(is_rand==true)
	{
		type="gauss";
	}
	else
	{
		type="ofirst";
	}
}

void Ensemble_param::print_to_console()
{
	std::cout << name <<": N="<< N << " mid="<< mid << " sigma=" << sigma << " " << type<<std::endl;
	std::cout << "   ";
	for (int i = 0; i < container.size(); ++i)
	{
		std::cout << container[i] << " ";
	}
	std::cout << std::endl;
}

prec& Ensemble_param::operator[](int i)
{
	return container[i];
}