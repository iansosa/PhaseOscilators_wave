#include "Ensemble_param.h"


void Ensemble_param::dist(prec in_mid, prec in_sigma)
{
	mid=in_mid;
	sigma=in_sigma;
}

int Ensemble_param::size()
{
	return N;
}

void Ensemble_param::resize(int in_N)
{
	N=in_N;
	container.resize(in_N);
	set_rand(false);
}

void Ensemble_param::set_rand(bool first)
{
	boost::normal_distribution<> gauss(mid,sigma);
    boost::variate_generator< boost::mt19937&, boost::normal_distribution<> > gen(rng,gauss);
    for (int i = 0; i < container.size(); ++i)
    {
    	container[i]=gen();
    }

    if(first==false)
    {
    	container[0]=mid;
    }
}

void Ensemble_param::print()
{
	std::ofstream txtOut;
	txtOut.open(name+".txt");
	txtOut.precision(8);
	txtOut << N << " ";
	txtOut << mid << " ";
	txtOut << sigma << " ";
	for (int i = 0; i < container.size(); ++i)
	{
		txtOut << container[i] << " ";
	}
	txtOut.close();
}

void Ensemble_param::load()
{
	std::ifstream txtIn_check;
	int N_check;
	txtIn_check.open("params.txt");
	txtIn_check >> N_check;
	txtIn_check.close();	
	std::ifstream txtIn;
	txtIn.open(name+".txt");
	txtIn >> N;
	txtIn >> mid;
	txtIn >> sigma;
	container.resize(N);
	if(N_check!=N)
	{
		std::cout << "INCONSISTENT PARAM SIZE FOR " << name << ". Expects N=" << N << " but check throws N=" << N_check <<std::endl;
	}
	for (int i = 0; i < container.size(); ++i)
	{
		txtIn >> container[i];
	}
	txtIn.close();
}

void Ensemble_param::print_to_console()
{
	for (int i = 0; i < container.size(); ++i)
	{
		std::cout << container[i] << " ";
	}
	std::cout << std::endl;
}

prec& Ensemble_param::operator[](int i)
{
	return container[i];
}