#include <string>
#include <vector>
#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <cstring>

#include <sys/stat.h>
#include <boost/numeric/odeint.hpp>
#include <boost/random.hpp>

typedef double prec;

#include "Ensemble_param.h"

void Ensemble_param::assign(int i, prec value)
{
	container[i]=value;
}

void Ensemble_param::pass_ref(std::vector<prec> &x)
{
	container=x;
}

void Ensemble_param::dist(prec in_mid, prec in_sigma)
{
	mid=in_mid;
	sigma=in_sigma;
}

int Ensemble_param::size()
{
	return N;
}

void Ensemble_param::resize(int in_N)
{
	N=in_N;
	container.resize(in_N);
	generate();
}

prec& Ensemble_param::operator[](int i)
{
	return container[i];
}

void Ensemble_param::generate()
{
	boost::normal_distribution<> gauss(mid,sigma);
    boost::variate_generator< boost::mt19937&, boost::normal_distribution<> > gen(rng,gauss);
    for (int i = 0; i < container.size(); ++i)
    {
    	container[i]=gen();
    }

    if(f_belong==false)
    {
    	container[0]=mid;
    }

    if(type=="ofirst")
    {
    	container[0]=mid;
    	for (int i = 1; i < container.size(); ++i)
    	{
    		container[i]=0;
    	}
    }
}

void Ensemble_param::print()
{
	mkdir("params", 0777);

	std::ofstream txtOut;
	txtOut.open("params/"+name+".txt");
	txtOut.precision(std::numeric_limits< prec >::max_digits10);
	txtOut << N << " ";
	txtOut << mid << " ";
	txtOut << sigma << " ";
	txtOut << f_belong << " ";
	txtOut << type << " ";
	
	for (int i = 0; i < container.size(); ++i)
	{
		txtOut << container[i] << " ";
	}
	txtOut.close();
}

void Ensemble_param::load()
{
	std::ifstream txtIn_check;
	int N_check;
	txtIn_check.open("params.txt");
	txtIn_check >> N_check;
	txtIn_check.close();	
	std::ifstream txtIn;
	txtIn.open("params/"+name+".txt");
	txtIn >> N;
	txtIn >> mid;
	txtIn >> sigma;
	txtIn >> f_belong;
	txtIn >> type;

	container.resize(N);
	if(N_check!=N)
	{
		std::cout << "INCONSISTENT PARAM SIZE FOR " << name << ". Expects N=" << N << " but check throws N=" << N_check <<std::endl;
	}
	for (int i = 0; i < container.size(); ++i)
	{
		txtIn >> container[i];
	}
	txtIn.close();
}

void Ensemble_param::type_rand(bool is_rand, bool c_f_belong)
{
	if(is_rand==true)
	{
		type="gauss";
	}
	else
	{
		type="ofirst";
	}
	f_belong=c_f_belong;
}

void Ensemble_param::print_to_console()
{
	std::cout << name <<": N="<< N << " mid="<< mid << " sigma=" << sigma << " " << type<<std::endl;
	std::cout << "   ";
	for (int i = 0; i < container.size(); ++i)
	{
		std::cout << container[i] << " ";
	}
	std::cout << std::endl;
}