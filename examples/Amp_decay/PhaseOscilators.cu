#include "hip/hip_runtime.h"
#include <string>
#include <vector>
#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <cstring>
#include <limits>

#include <sys/stat.h>
#include <boost/numeric/odeint.hpp>
#include <boost/numeric/odeint/external/openmp/openmp.hpp>
#include <boost/random.hpp>

typedef double prec;

#include "../../Ensemble_param.h"
#include "../../Ensemble_connections.h"
#include "../../Ensemble_Dinamic.h"
#include "../../Ensemble_evolve.h"

void calc_U(Evolve &e, Dinamic &P, prec dx, std::vector<prec> &U)
{
	for (int i = 1; i < U.size(); ++i)
	{
		prec aux=e.Amp(i,P);
		if(aux>0)
		{
			U[i]=log(aux);
		}
	}

	for (int i = 1; i < U.size()-1; ++i)
	{
		U[i]=(U[i+1]-U[i])/dx;
	}
}

void calc_U_RMS(Evolve &e, Dinamic &P, prec dx, std::vector<prec> &U)
{
	for (int i = 1; i < U.size(); ++i)
	{
		prec aux=sqrt(2)*e.RMS(i,P);
		if(aux>0)
		{
			U[i]=log(aux);
		}
	}

	for (int i = 1; i < U.size()-1; ++i)
	{
		U[i]=(U[i+1]-U[i])/dx;
	}
}

void printU(Evolve &e, Dinamic &P, prec dx,prec I, prec G,int N)
{
	std::vector<std::vector <prec>> U(2);
	for (int i = 0; i < U.size(); ++i)
	{
		U[i].resize(e.size(),0);
	}
	//std::vector<prec> U(e.size(),0);

	std::cout << "U..." << std::endl;
	calc_U(e,P,dx,U[0]);
	std::cout << "RMS U..." << std::endl;
	calc_U_RMS(e,P,dx,U[1]);

	std::string name="U";
	mkdir("out", 0777);
	std::ofstream txtOut;
	txtOut.open("out/"+name+"_"+std::to_string(I)+"_"+std::to_string(G)+"_"+std::to_string(N)+".txt");
	txtOut.precision(std::numeric_limits< prec >::max_digits10);

	std::cout << "Printing U..." << std::endl;
	for (int i = 1; i < U[0].size()-1; ++i)
	{
			txtOut << i << " " << U[0][i] << " " << U[1][i] << " " << e.Diff_max(i) << " " << e.Diff_med(i) << " " << e.Diff_min(i) << std::endl;
	}
	txtOut.close();	
}

void print_U_stats(std::vector<std::vector<prec>> &U)
{
	std::vector<prec> U_med(U[0].size(),0);
	std::vector<prec> U_sigm(U[0].size(),0);
	for (int i = 0; i < U[0].size(); ++i) //N
	{
		int norma=0;
		for (int j = 0; j < U.size(); ++j) //promedio
		{
			if(U[j][i]>-1000 && U[j][i]<500)
			{
				norma++;
				U_med[i]=U_med[i]+U[j][i];
			}	
		}
		if(norma>0)
		{
			U_med[i]=U_med[i]/norma;
		}
	}

	for (int i = 0; i < U[0].size(); ++i) //N
	{
		int norma=0;
		for (int j = 0; j < U.size(); ++j) //promedio
		{
			if(U[j][i]>-1000 && U[j][i]<500)
			{
				norma++;
				U_sigm[i]=pow(U[j][i]-U_med[i],2);
			}	
		}
		if(norma>0)
		{
			U_sigm[i]=U_sigm[i]/norma;
		}
		U_sigm[i]=sqrt(U_sigm[i]);
	}
	std::string name="U_stats";
	mkdir("out", 0777);
	std::ofstream txtOut;
	txtOut.open("out/"+name+".txt");
	txtOut.precision(std::numeric_limits< prec >::max_digits10);

	for (int i = 0; i < U_med.size(); ++i)
	{
			txtOut << i << " " << U_med[i] << " " << U_sigm[i] << std::endl;
	}
	txtOut.close();	
}

void promedio_U()
{
 	boost::mt19937 rng(static_cast<unsigned int>(std::time(0)));

 	int N;
 	std::cout << "N: ";
 	std::cin >> N;

  	int prom;
 	std::cout << "Prom: ";
 	std::cin >> prom;

 	std::vector<std::vector<prec> > U(prom);

 	int nProcessors=omp_get_max_threads();

    int chunk_size = N/nProcessors;

    omp_set_num_threads(nProcessors);

    omp_set_schedule( omp_sched_static , chunk_size );

    prec conv_factor=5;
    prec dx=0.01;
    dx=dx/conv_factor;

    prec K=conv_factor*conv_factor*N;
    prec I=0.1;
    prec G=5;
	Dinamic P(N ,K ,rng ,I ,I*0.01 ,1 ,0 ,G ,G*0.01 ,1 ,0,"c_a_chain"); //N, K, rng, I, sigm_I, F, sigm_F, G, sigm_G, W, sigm_W
	P.init_I_type_rand(true,false);

	Evolve e(N, rng);
	for (int i = 0; i < prom; ++i)
	{
		std::cout << "Iter: (" << i+1 << "/" << prom << ")" << std::endl;
		U[i].resize(N);
		e.run(P,0,50000,100);
		e.calc_convergence();
		calc_U(e,P,dx,U[i]);
		e.clean();
		P.print_params();

		P.generate();
	}
	e.print();

	print_U_stats(U);
}


int main()
{

	return 0;
}