#include "hip/hip_runtime.h"
#include <string>
#include <vector>
#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <cstring>

#include <sys/stat.h>
#include <boost/numeric/odeint.hpp>
#include <boost/numeric/odeint/external/openmp/openmp.hpp>
#include <boost/random.hpp>

typedef double prec;

#include "Ensemble_param.h"
#include "Ensemble_connections.h"
#include "Ensemble_Dinamic.h"


prec Dinamic::to_odeint::interaction_sum_all(int id, const std::vector<prec> &x)
{
	prec sum=0;
	for (int i = 0; i < N_t_odeint; ++i)
	{
		sum=sum+A_t_odeint(id,i)*sin(x[2*i]-x[2*id])/N_t_odeint;
	}
	return sum*K_t_odeint;
}

prec Dinamic::to_odeint::interaction_sum_chain(int id,const std::vector<prec> &x)
{
			prec sum=0;
		    if(id>0 && id<N_t_odeint-1)
		    {
		    	sum=sum+A_t_odeint(id,id+1)*sin(x[2*(id+1)]-x[2*id])/N_t_odeint;
		    	sum=sum+A_t_odeint(id,id-1)*sin(x[2*(id-1)]-x[2*id])/N_t_odeint;
		    	return sum*K_t_odeint;
 		    }
 		    if(id==0)
 		    {
		    	sum=sum+A_t_odeint(id,id+1)*sin(x[2*(id+1)]-x[2*id])/N_t_odeint;
		    	return sum*K_t_odeint;
		    }
		    if(id==N_t_odeint-1)
		    {
		    	sum=sum+A_t_odeint(id,id-1)*sin(x[2*(id-1)]-x[2*id])/N_t_odeint;
		    	return sum*K_t_odeint;
		    }  
			return sum*K_t_odeint;
}

prec Dinamic::to_odeint::interaction_sum(int id, const std::vector<prec> &x)
{
	if(A_t_odeint.get_type()=="chain")
	{
		return interaction_sum_chain(id,x);
	}
	if(A_t_odeint.get_type()=="global" || A_t_odeint.get_type()=="custom")
	{
		return interaction_sum_all(id,x);
	}
}


void Dinamic::print_params()
{
	std::ofstream txtOut;
	txtOut.open("params.txt");
	txtOut.precision(8);
	txtOut << N << " " << K << std::endl;
	txtOut.close();
	A.print();
   	I.print();
   	F.print();
   	G.print();
   	W.print();
}

void Dinamic::print_params_to_console()
{
	A.print_to_console();
	I.print_to_console();
	F.print_to_console();
	G.print_to_console();
	W.print_to_console();
}

void Dinamic::generate()
{
	A.generate(A.get_type());
   	I.generate();
   	F.generate();
   	G.generate();
   	W.generate();
}