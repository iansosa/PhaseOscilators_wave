#include "hip/hip_runtime.h"
#include <string>
#include <vector>
#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <cstring>

#include <sys/stat.h>
#include <boost/numeric/odeint.hpp>
#include <boost/numeric/odeint/external/openmp/openmp.hpp>
#include <boost/random.hpp>

typedef double prec;

#include "Ensemble_param.h"
#include "Ensemble_connections.h"
#include "Ensemble_Dinamic.h"

prec Dinamic::to_odeint::interaction_sum_chain(int id,const std::vector<prec> &x)
{
			prec sum=0;
		    if(id>0 && id<N_t_odeint-1)
		    {
		    	sum=sum+A_t_odeint(id,id+1)*sin(x[2*(id+1)]-x[2*id])/N_t_odeint;
		    	sum=sum+A_t_odeint(id,id-1)*sin(x[2*(id-1)]-x[2*id])/N_t_odeint;
		    	return sum*K_t_odeint;
 		    }
 		    if(id==0)
 		    {
		    	sum=sum+A_t_odeint(id,id+1)*sin(x[2*(id+1)]-x[2*id])/N_t_odeint;
		    	return sum*K_t_odeint;
		    }
		    if(id==N_t_odeint-1)
		    {
		    	sum=sum+A_t_odeint(id,id-1)*sin(x[2*(id-1)]-x[2*id])/N_t_odeint;
		    	return sum*K_t_odeint;
		    }  
			return sum*K_t_odeint;
}

prec Dinamic::to_odeint::h_interaction_sum_chain(int id,const std::vector<prec> &x)
{
			prec sum=0;
		    if(id>0 && id<N_t_odeint-1)
		    {
		    	sum=sum+A_t_odeint(id,id+1)*(x[2*(id+1)]-x[2*id])/N_t_odeint;
		    	sum=sum+A_t_odeint(id,id-1)*(x[2*(id-1)]-x[2*id])/N_t_odeint;
		    	return sum*K_t_odeint;
 		    }
 		    if(id==0)
 		    {
		    	sum=sum+A_t_odeint(id,id+1)*(x[2*(id+1)]-x[2*id])/N_t_odeint;
		    	return sum*K_t_odeint;
		    }
		    if(id==N_t_odeint-1)
		    {
		    	sum=sum+A_t_odeint(id,id-1)*(x[2*(id-1)]-x[2*id])/N_t_odeint;
		    	return sum*K_t_odeint;
		    }  
			return sum*K_t_odeint;
}

prec Dinamic::to_odeint::interaction_sum(int id, const std::vector<prec> &x)
{
	if(A_t_odeint.get_type()=="chain")
	{
		return interaction_sum_chain(id,x);
	}
	if(A_t_odeint.get_type()=="global" || A_t_odeint.get_type()=="custom")
	{
		return interaction_sum_all(id,x);
	}
	if(A_t_odeint.get_type()=="h_chain")
	{
		return h_interaction_sum_chain(id,x);
	}
	return 0;
}

prec Dinamic::to_odeint::force_sum(prec sum, int id, const std::vector<prec> &x,const prec t)
{
	if(A_t_odeint.get_type()=="chain" || A_t_odeint.get_type()=="global" || A_t_odeint.get_type()=="custom")
	{
		return p_force_sum(sum,id,x,t);
	}
	if(A_t_odeint.get_type()=="h_chain")
	{
		return h_force_sum(sum,id,x,t);
	}
	return 0;
}

prec Dinamic::to_odeint::h_force_sum(prec sum, int i,const std::vector<prec> &x,const prec t)
{
			return sum/I_t_odeint[i]+F_t_odeint[i]*(sin(W_t_odeint[i]*t)-x[2*i])/I_t_odeint[i]-(G_t_odeint[i]/I_t_odeint[i])*x[2*i+1];
}

prec Dinamic::to_odeint::p_force_sum(prec sum, int i,const std::vector<prec> &x,const prec t)
{
			return sum/I_t_odeint[i]+F_t_odeint[i]*sin(W_t_odeint[i]*t-x[2*i])/I_t_odeint[i]-(G_t_odeint[i]/I_t_odeint[i])*x[2*i+1];
}

prec Dinamic::to_odeint::interaction_sum_all(int id, const std::vector<prec> &x)
{
	prec sum=0;
	for (int i = 0; i < N_t_odeint; ++i)
	{
		sum=sum+A_t_odeint(id,i)*sin(x[2*i]-x[2*id])/N_t_odeint;
	}
	return sum*K_t_odeint;
}

void Dinamic::print_params()
{
	std::ofstream txtOut;
	txtOut.open("params.txt");
	txtOut.precision(8);
	txtOut << N << " " << K << std::endl;
	txtOut.close();
	A.print();
   	I.print();
   	F.print();
   	G.print();
   	W.print();
}

void Dinamic::print_params_to_console()
{
	A.print_to_console();
	I.print_to_console();
	F.print_to_console();
	G.print_to_console();
	W.print_to_console();
}

void Dinamic::generate()
{
	A.generate(A.get_type());
   	I.generate();
   	F.generate();
   	G.generate();
   	W.generate();
}

std::string Dinamic::get_type()
{
	return A.get_type();
}