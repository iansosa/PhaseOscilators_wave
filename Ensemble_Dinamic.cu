#include "hip/hip_runtime.h"
#include <string>
#include <vector>
#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <cstring>

#include <sys/stat.h>
#include <boost/numeric/odeint.hpp>
#include <boost/numeric/odeint/external/openmp/openmp.hpp>
#include <boost/random.hpp>

typedef long double prec;

#include "Ensemble_param.h"
#include "Ensemble_connections.h"
#include "Ensemble_Dinamic.h"

prec Dinamic::to_odeint::interaction_sum_chain(int id,const std::vector<prec> &x)
{
			prec sum=0;
		    if(id>0 && id<N_t_odeint-1)
		    {
		    	sum=sum+A_t_odeint(id,id+1)*sin(x[2*(id+1)]-x[2*id])/N_t_odeint;
		    	sum=sum+A_t_odeint(id,id-1)*sin(x[2*(id-1)]-x[2*id])/N_t_odeint;
		    	return sum*K_t_odeint;
 		    }
 		    if(id==0)
 		    {
		    	sum=sum+A_t_odeint(id,id+1)*sin(x[2*(id+1)]-x[2*id])/N_t_odeint;
		    	return sum*K_t_odeint;
		    }
		    if(id==N_t_odeint-1)
		    {
		    	sum=sum+A_t_odeint(id,id-1)*sin(x[2*(id-1)]-x[2*id])/N_t_odeint;
		    	return sum*K_t_odeint;
		    }  
			return sum*K_t_odeint;
}

prec Dinamic::to_odeint::h_interaction_sum_chain(int id,const std::vector<prec> &x)
{
			prec sum=0;
		    if(id>0 && id<N_t_odeint-1)
		    {

		    	sum=sum+A_t_odeint(id,id+1)*(x[2*(id+1)]-x[2*id])/N_t_odeint;
		    	sum=sum+A_t_odeint(id,id-1)*(x[2*(id-1)]-x[2*id])/N_t_odeint;		    	
		    	return sum*K_t_odeint;
 		    }
 		    if(id==0)
 		    {
		    	sum=sum+A_t_odeint(id,id+1)*(x[2*(id+1)]-x[2*id])/N_t_odeint;
		    	return sum*K_t_odeint;
		    }
		    if(id==N_t_odeint-1)
		    {
		    	sum=sum+A_t_odeint(id,id-1)*(x[2*(id-1)]-x[2*id])/N_t_odeint;
		    	return sum*K_t_odeint;
		    }  
			return sum*K_t_odeint;
}

prec Dinamic::to_odeint::interaction_sum(int id, const std::vector<prec> &x)
{
	switch(A_t_odeint.type_id)
	{
		case 0:
			return interaction_sum_all(id,x);
			break;
		case 1:
			return interaction_sum_chain(id,x);
			break;
		case 2:
			return interaction_sum_all(id,x);
			break;
		case 3:
			return h_interaction_sum_chain(id,x);
			break;
		case 4:
			return h_interaction_sum_chain(id,x);
			break;
		case 5:
			return h_interaction_sum_chain(id,x);
			break;
		case 6:
			return h_interaction_sum_chain(id,x);
			break;
		case 7:
			return 0;
			break;
		default :
			std::cout << "undefined ensemble type" << std::endl;
			return 0;
			break;
	}
}

prec Dinamic::to_odeint::force_sum(prec sum, int id, const std::vector<prec> &x,const prec t)
{
	switch(A_t_odeint.type_id)
	{
		case 0:
			return p_force_sum(sum,id,x,t);
			break;
		case 1:
			return p_force_sum(sum,id,x,t);
			break;
		case 2:
			return p_force_sum(sum,id,x,t);
			break;
		case 3:
			return h_force_sum(sum,id,x,t);
			break;
		case 4:
			return t_force_sum(sum,id,x,t);
			break;
		case 5:
			return s_force_sum(sum,id,x,t);
			break;
		case 6:
			return c_a_force_sum(sum,id,x,t);
			break;
		case 7:
			return solid_force_sum(id,x,t);
			break;
		default :
			std::cout << "undefined ensemble type" << std::endl;
			return 0;
			break;
	}
}

prec Dinamic::to_odeint::p_force_sum(prec sum, int i,const std::vector<prec> &x,const prec t)
{
	return sum/I_t_odeint[i]+F_t_odeint[i]*sin(W_t_odeint[i]*t-x[2*i])/I_t_odeint[i]-(G_t_odeint[i]/I_t_odeint[i])*x[2*i+1];
}

prec Dinamic::to_odeint::h_force_sum(prec sum, int i,const std::vector<prec> &x,const prec t)
{
	return sum/I_t_odeint[i]+F_t_odeint[i]*(sin(W_t_odeint[i]*t)-x[2*i])/I_t_odeint[i]-(G_t_odeint[i]/I_t_odeint[i])*x[2*i+1];
}

prec Dinamic::to_odeint::t_force_sum(prec sum, int i,const std::vector<prec> &x,const prec t)
{
	if(F_t_odeint[i]<0.0000001 && F_t_odeint[i]>-0.0000001)
	{
		return sum/I_t_odeint[i]-(G_t_odeint[i]/I_t_odeint[i])*x[2*i+1];	
	}
	else
	{
		int n=static_cast<int>(t*W_t_odeint[i]/(2*M_PI));
		prec fixed_t=t-n*2.0*M_PI/W_t_odeint[i];

		if(fixed_t<1.0*M_PI/W_t_odeint[i])
		{
			return sum/I_t_odeint[i]+F_t_odeint[i]*(-1+2.0*fixed_t/(1.0*M_PI/W_t_odeint[i]) -x[2*i])/I_t_odeint[i]-(G_t_odeint[i]/I_t_odeint[i])*x[2*i+1];
		}
		else
		{
			return sum/I_t_odeint[i]+F_t_odeint[i]*(1-2.0*(fixed_t-1.0*M_PI/W_t_odeint[i])/(1.0*M_PI/W_t_odeint[i]) -x[2*i])/I_t_odeint[i]-(G_t_odeint[i]/I_t_odeint[i])*x[2*i+1];
		}
	}
}

prec Dinamic::to_odeint::s_force_sum(prec sum, int i,const std::vector<prec> &x,const prec t)
{
	if(F_t_odeint[i]<0.0000001 && F_t_odeint[i]>-0.0000001)
	{
		return sum/I_t_odeint[i]-(G_t_odeint[i]/I_t_odeint[i])*x[2*i+1];
	}
	else
	{
		int n=static_cast<int>(t*W_t_odeint[i]/(2*M_PI));
		prec fixed_t=t-n*2.0*M_PI/W_t_odeint[i];

		return sum/I_t_odeint[i]+F_t_odeint[i]*(-1+2.0*fixed_t/(2.0*M_PI/W_t_odeint[i]) -x[2*i])/I_t_odeint[i]-(G_t_odeint[i]/I_t_odeint[i])*x[2*i+1];
	}
}

prec Dinamic::to_odeint::c_a_force_sum(prec sum, int i,const std::vector<prec> &x,const prec t)
{
//return sum/I_t_odeint[i]+F_t_odeint[i]*(sin(W_t_odeint[i]*t)+(1.0/5.0)*sin(2*W_t_odeint[i]*t)-(1.0/5.0)*sin(3*W_t_odeint[i]*t)-x[2*i])/I_t_odeint[i]-(G_t_odeint[i]/I_t_odeint[i])*x[2*i+1];
	//F.assign(0,25);
	return sum/I_t_odeint[i]+F_t_odeint[i]*(sin(W_t_odeint[i]*t)+(0.473916/2.0)*sin(2*W_t_odeint[i]*t)+(0.221962/3.0)*sin(3*W_t_odeint[i]*t)+(0.102042/4.0)*sin(4*W_t_odeint[i]*t)-x[2*i])/I_t_odeint[i]-(G_t_odeint[i]/I_t_odeint[i])*x[2*i+1];
}

prec Dinamic::to_odeint::solid_force_sum(int i,const std::vector<prec> &x,const prec t)
{
	return -K_t_odeint*sin(x[2*i])/I_t_odeint[i]+F_t_odeint[i]*sin(W_t_odeint[i]*t-x[2*i])/I_t_odeint[i]-(G_t_odeint[i]/I_t_odeint[i])*x[2*i+1];
}


prec Dinamic::to_odeint::interaction_sum_all(int id, const std::vector<prec> &x)
{
	prec sum=0;
	for (int i = 0; i < N_t_odeint; ++i)
	{
		sum=sum+A_t_odeint(id,i)*sin(x[2*i]-x[2*id])/N_t_odeint;
	}
	return sum*K_t_odeint;
}

void Dinamic::print_params()
{
	std::ofstream txtOut;
	txtOut.open("params.txt");
	txtOut.precision(8);
	txtOut << N << " " << K << std::endl;
	txtOut.close();
	A.print();
   	I.print();
   	F.print();
   	G.print();
   	W.print();
}

void Dinamic::print_params_to_console()
{
	A.print_to_console();
	I.print_to_console();
	F.print_to_console();
	G.print_to_console();
	W.print_to_console();
}

void Dinamic::generate()
{
	A.generate(A.get_type());
   	I.generate();
   	F.generate();
   	G.generate();
   	W.generate();
}

void Dinamic::init_I_type_rand(bool c_bool, bool c_f_belong)
{
	I.type_rand(c_bool, c_f_belong);
	I.generate();
}

void Dinamic::init_F_type_rand(bool c_bool, bool c_f_belong)
{
	F.type_rand(c_bool, c_f_belong);
	F.generate();
}

void Dinamic::init_G_type_rand(bool c_bool, bool c_f_belong)
{
	G.type_rand(c_bool, c_f_belong);
	G.generate();
}

void Dinamic::init_W_type_rand(bool c_bool, bool c_f_belong)
{
	W.type_rand(c_bool, c_f_belong);
	W.generate();
}

std::string Dinamic::get_type()
{
	return A.get_type();
}

void Dinamic::new_va_F(int id,prec val)
{
	F.assign(id,val);
}
