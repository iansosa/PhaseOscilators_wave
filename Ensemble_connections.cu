#include <string>
#include <vector>
#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <cstring>

#include <sys/stat.h>
#include <boost/numeric/odeint.hpp>
#include <boost/random.hpp>

typedef double prec;

#include "Ensemble_connections.h"

int Ensemble_connections::size()
{
	return N;
}

void Ensemble_connections::resize(int i)
{
	N=i;
	container.resize(N*N);
	generate(type);
}

void Ensemble_connections::generate(std::string s)
{
	if(s=="global")
	{
		std::fill(container.begin(), container.end(), 1);
	}
	if(s=="empty")
	{
		std::fill(container.begin(), container.end(), 0);
	}
	if(s=="chain")
	{
		std::fill(container.begin(), container.end(), 0);
		for (int i = 0; i < N-1; ++i)
		{
			connect(i,i+1);
		}
	}
	if(s=="h_chain")
	{
		std::fill(container.begin(), container.end(), 0);
		for (int i = 0; i < N-1; ++i)
		{
			connect(i,i+1);
		}
	}
}

void Ensemble_connections::connect(int i, int j)
{
	container[_2dto1d(i, j)]=1;
	container[_2dto1d(j, i)]=1;
}

void Ensemble_connections::print()
{
	mkdir("params", 0777);

	std::ofstream txtOut;
	txtOut.open("params/"+name+".txt");
	txtOut.precision(8);
	txtOut << N << " ";
	txtOut << k << " ";
	txtOut << proba << " ";
	txtOut << type << " ";
	for (int i = 0; i < container.size(); ++i)
	{
		txtOut << container[i] << " ";
	}
	txtOut.close();
}

void Ensemble_connections::load()
{
	std::ifstream txtIn_check;
	int N_check;
	txtIn_check.open("params.txt");
	txtIn_check >> N_check;
	txtIn_check.close();	
	std::ifstream txtIn;
	txtIn.open("params/"+name+".txt");
	txtIn >> N;
	txtIn >> k;
	txtIn >> proba;
	txtIn >> type;
	container.resize(N*N);
	if(N_check!=N)
	{
		std::cout << "INCONSISTENT PARAM SIZE FOR " << name << ". Expects N=" << N << " but check throws N=" << N_check <<std::endl;
	}
	for (int i = 0; i < container.size(); ++i)
	{
		txtIn >> container[i];
	}
	txtIn.close();
}

std::string Ensemble_connections::get_type()
{
	return type;
}

void Ensemble_connections::print_to_console()
{
	std::cout << name <<": N="<< N << " k="<< k << " proba=" << proba << " " << type<<std::endl;
	for (int i = 0; i < N; ++i)
	{
		std::cout << "   ";
		for (int j = 0; j < N; ++j)
		{
			std::cout << container[_2dto1d(i, j)] << " ";
		}

		std::cout << std::endl;
	}
}

int Ensemble_connections::_2dto1d(int a, int b)
{
	return a+b*N;
}

void Ensemble_connections::_1dto2d(int idx,int &a, int &b)
{
	a=idx%N;
	b=idx/N;
}