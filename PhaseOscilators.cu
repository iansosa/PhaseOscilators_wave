#include "hip/hip_runtime.h"
#include <string>
#include <vector>
#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <cstring>
#include <limits>

#include <sys/stat.h>
#include <boost/numeric/odeint.hpp>
#include <boost/numeric/odeint/external/openmp/openmp.hpp>
#include <boost/random.hpp>

typedef double prec;

#include "Ensemble_param.h"
#include "Ensemble_connections.h"
#include "Ensemble_Dinamic.h"
#include "Ensemble_evolve.h"

void check_stationary(Evolve &e, Dinamic &P)
{
	bool statement=false;
	int count=0;
	while(statement==false)
	{
		e.clean();
		e.run(P,count*10000,(count+1)*10000,100);

		for (int i = 1; i < e.size(); ++i)
		{	
			prec convergence=e.convergence(i);
			if(convergence>0)
			{
				std::cout << i <<": " << convergence << std::endl;
			}
		}

		statement=e.did_converge(49);
		count++;

		e.reset_init();		
	}
	e.print();
	e.print_init();
}

void calc_U(Evolve &e, Dinamic &P, prec dx, std::vector<prec> &U)
{
	for (int i = 1; i < U.size(); ++i)
	{
		prec aux=e.Amp(i,P);
		if(aux>0)
		{
			U[i]=log(aux);
		}
	}

	for (int i = 1; i < U.size()-1; ++i)
	{
		U[i]=(U[i+1]-U[i])/dx;
	}
}

void printU(Evolve &e, Dinamic &P, prec dx)
{
	std::vector<prec> U(e.size(),0);

	calc_U(e,P,dx,U);

	std::string name="U";
	mkdir("out", 0777);
	std::ofstream txtOut;
	txtOut.open("out/"+name+".txt");
	txtOut.precision(std::numeric_limits< prec >::max_digits10);

	for (int i = 1; i < U.size()-1; ++i)
	{
			txtOut << i << " " << U[i] << std::endl;
	}
	txtOut.close();	
}

void print_U_stats(std::vector<std::vector<prec>> &U)
{
	std::vector<prec> U_med(U[0].size(),0);
	std::vector<prec> U_sigm(U[0].size(),0);
	for (int i = 0; i < U[0].size(); ++i) //N
	{
		for (int j = 0; j < U.size(); ++j) //promedio
		{
			U_med[i]=U_med[i]+U[j][i];
		}
		U_med[i]=U_med[i]/U.size();
	}

	for (int i = 0; i < U[0].size(); ++i) //N
	{
		for (int j = 0; j < U.size(); ++j) //promedio
		{
			U_sigm[i]=pow(U[j][i]-U_med[i],2);
		}
		U_sigm[i]=U_sigm[i]/U.size();
		U_sigm[i]=sqrt(U_sigm[i]);
	}
	std::string name="U_stats";
	mkdir("out", 0777);
	std::ofstream txtOut;
	txtOut.open("out/"+name+".txt");
	txtOut.precision(std::numeric_limits< prec >::max_digits10);

	for (int i = 0; i < U_med.size(); ++i)
	{
			txtOut << i << " " << U_med[i] << " " << U_sigm[i] << std::endl;
	}
	txtOut.close();	
}

int main()
{
 	boost::mt19937 rng(static_cast<unsigned int>(std::time(0)));

 	int N;
 	std::cout << "N: ";
 	std::cin >> N;

  	int prom;
 	std::cout << "Prom: ";
 	std::cin >> prom;

 	std::vector<std::vector<prec> > U(prom);

    int chunk_size = N/omp_get_max_threads();
    omp_set_schedule( omp_sched_static , chunk_size );

    prec conv_factor=5;
    prec dx=0.01;
    dx=dx/conv_factor;

    prec K=conv_factor*conv_factor*N;
    prec I=0.1;
    prec G=5;
	Dinamic P(N ,K ,rng ,I ,I*0.01 ,1 ,0 ,G ,G*0.01 ,1 ,0,"chain"); //N, K, rng, I, sigm_I, F, sigm_F, G, sigm_G, W, sigm_W

	Evolve e(N, rng);
	for (int i = 0; i < prom; ++i)
	{
		std::cout << "Iter: (" << i+1 << "/" << prom << ")" << std::endl;
		U[i].resize(N);
		e.run(P,0,50000,100);
		calc_U(e,P,dx,U[i]);
		e.clean();

		P.generate();
	}

	print_U_stats(U);

	return 0;
}