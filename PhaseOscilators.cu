#include "hip/hip_runtime.h"
#include <string>
#include <vector>
#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <cstring>

#include <boost/numeric/odeint.hpp>
#include <boost/random.hpp>

typedef double prec;

class Ensemble_param
{
	int N;
	std::string name;
	boost::mt19937 *rng;

	public:
		std::vector<prec> container;
		prec mid;
		prec sigma;
		Ensemble_param(int in_N, std::string in_name, prec in_mid, prec in_sigma, boost::mt19937 &in_rng) : N(in_N), name(in_name), rng(&in_rng), mid(in_mid), sigma(in_sigma), container(N)
		{
			set();
		}

		Ensemble_param(int in_N, std::string in_name) : N(in_N), name(in_name), container(N), rng(NULL)
		{
			load();
		}

		prec& operator[](int i)
		{
			return container[i];
		}

		void set()
		{
			boost::normal_distribution<> gauss(mid,sigma);
    		boost::variate_generator< boost::mt19937&, boost::normal_distribution<> > gen(*rng,gauss);

    		for (int i = 0; i < N; ++i)
    		{
    			container[i]=gen();
    		}
		}

		void print()
		{
			std::ofstream txtOut;
			txtOut.open(name+".txt");
			txtOut.precision(8);
			for (int i = 0; i < N; ++i)
			{
				txtOut << container[i] << " ";
			}
			txtOut.close();
		}

		void load()
		{
			std::ifstream txtIn;
			txtIn.open(name+".txt");
			for (int i = 0; i < N; ++i)
			{
				txtIn >> container[i];
			}
			txtIn.close();
		}

		void print_to_console()
		{
			for (int i = 0; i < N; ++i)
			{
				std::cout << container[i] << " ";
			}
			std::cout << std::endl;
		}

};

class Dinamic 
{
    prec K;

    Ensemble_param I;
    Ensemble_param F;
    std::vector<std::vector<prec>> A;
    Ensemble_param G;
    Ensemble_param W;

	public:
		int N;

    	Dinamic(int in_N , prec in_K) : N(in_N), K(in_K), I(in_N,"I"), F(in_N,"F"), A(in_N), G(in_N,"G"), W(in_N,"W")
    	{ }

    	Dinamic(int in_N , prec in_K, boost::mt19937 &in_rng, prec mid_I, prec sigma_I, prec mid_F, prec sigma_F, prec mid_G, prec sigma_G, prec mid_W, prec sigma_W) : N(in_N), K(in_K), I(in_N,"I",mid_I,sigma_I,in_rng), F(in_N,"F",mid_F,sigma_F,in_rng), A(in_N), G(in_N,"G",mid_G,sigma_G,in_rng), W(in_N,"W",mid_W,sigma_W,in_rng)
    	{ }

    	void operator() (const std::vector<prec> &x ,std::vector<prec> &dxdt ,const double t)
    	{
    	    for (int i = 0; i < N; ++i)
    	    {
    	    	prec sum=interaction_sum_chain(i,x);

        		dxdt[2*i]=x[2*i+1];
				dxdt[2*i+1]=sum/I[i]+F[i]*sin(W[i]*t-x[2*i])/I[i]-(G[i]/I[i])*x[2*i+1];      	
        	}
    	}

    	prec interaction_sum_all(int id, const std::vector<prec> &x)
    	{
    		prec sum=0;
    		for (int i = 0; i < N; ++i)
    		{
    			sum=sum+A[id][i]*sin(x[2*i]-x[2*id])/N;
    		}
    		return sum*K;
    	}

    	prec interaction_sum_chain(int id,const std::vector<prec> &x)
    	{
    		prec sum=0;
    	    if(id==0)
    	    {
    	    	sum=sum+A[id][id+1]*sin(x[2*(id+1)]-x[2*id])/N;
    	    }
    	    if(id==N-1)
    	    {
    	    	sum=sum+A[id][id-1]*sin(x[2*(id-1)]-x[2*id])/N;
    	    }
    	    if(id>0 && id<N-1)
    	    {
    	    	sum=sum+A[id][id+1]*sin(x[2*(id+1)]-x[2*id])/N;
    	    	sum=sum+A[id][id-1]*sin(x[2*(id-1)]-x[2*id])/N;
    	    }
    	   
    		return sum*K;
    	}

    	void print_params()
    	{
    		I.print();
    		F.print();
    		G.print();
    		W.print();
    	}

    	void print_params_to_console()
    	{
    		I.print_to_console();
    		F.print_to_console();
    		G.print_to_console();
    		W.print_to_console();
    	}
};

int main()
{
 	boost::mt19937 rng(static_cast<unsigned int>(std::time(0)));

	std::cout << "Hello World" << std::endl;


	Dinamic P(10, 1);
	P.print_params_to_console();

	return 0;
}