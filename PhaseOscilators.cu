#include "hip/hip_runtime.h"
#include <string>
#include <vector>
#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <cstring>

#include <boost/numeric/odeint.hpp>
#include <boost/random.hpp>

typedef double prec;

class Ensemble_param
{
	int N;
	std::string name;
	boost::mt19937 &rng;

	public:
		std::vector<prec> container;
		prec mid;
		prec sigma;
		Ensemble_param(int in_N, std::string in_name, prec in_mid, prec in_sigma, boost::mt19937 &in_rng) : N(in_N), name(in_name), rng(in_rng), mid(in_mid), sigma(in_sigma), container(N)
		{
			set();
		}

		Ensemble_param(std::string in_name, boost::mt19937 &in_rng) : name(in_name), rng(in_rng)
		{
			load();
		}

		prec& operator[](int i)
		{
			return container[i];
		}

		void set()
		{
			boost::normal_distribution<> gauss(mid,sigma);
    		boost::variate_generator< boost::mt19937&, boost::normal_distribution<> > gen(rng,gauss);

    		for (int i = 0; i < N; ++i)
    		{
    			container[i]=gen();
    		}
		}

		void print()
		{
			std::ofstream txtOut;
			txtOut.open(name+".txt");
			txtOut.precision(8);
			txtOut << N << " ";
			txtOut << mid << " ";
			txtOut << sigma << " ";
			for (int i = 0; i < container.size(); ++i)
			{
				txtOut << container[i] << " ";
			}
			txtOut.close();
		}

		void load()
		{
			std::ifstream txtIn_check;
			int N_check;
			txtIn_check.open("params.txt");
			txtIn_check >> N_check;
			txtIn_check.close();

			
			std::ifstream txtIn;
			txtIn.open(name+".txt");
			txtIn >> N;
			txtIn >> mid;
			txtIn >> sigma;
			container.resize(N);
			if(N_check!=N)
			{
				std::cout << "INCONSISTENT PARAM SIZE FOR " << name << ". Expects N=" << N << " but check throws N=" << N_check <<std::endl;
			}
			for (int i = 0; i < container.size(); ++i)
			{
				txtIn >> container[i];
			}
			txtIn.close();
		}

		void print_to_console()
		{
			for (int i = 0; i < container.size(); ++i)
			{
				std::cout << container[i] << " ";
			}
			std::cout << std::endl;
		}

};

class Dinamic 
{
    prec K;

    Ensemble_param I;
    Ensemble_param F;
    std::vector<std::vector<prec>> A;
    Ensemble_param G;
    Ensemble_param W;

    boost::mt19937 &rng;

	public:
		int N;

    	Dinamic(boost::mt19937 &in_rng) : rng(in_rng), I("I",in_rng), F("F",in_rng), G("G",in_rng), W("W",in_rng)
    	{
			std::ifstream txtIn_check;
			txtIn_check.open("params.txt");
			txtIn_check >> N;
			txtIn_check >> K;
			txtIn_check.close();
    	}

    	Dinamic(int in_N , prec in_K, boost::mt19937 &in_rng, prec mid_I, prec sigma_I, prec mid_F, prec sigma_F, prec mid_G, prec sigma_G, prec mid_W, prec sigma_W) : N(in_N), K(in_K), rng(in_rng), I(in_N,"I",mid_I,sigma_I,in_rng), F(in_N,"F",mid_F,sigma_F,in_rng), A(in_N), G(in_N,"G",mid_G,sigma_G,in_rng), W(in_N,"W",mid_W,sigma_W,in_rng)
    	{ }

    	void operator() (const std::vector<prec> &x ,std::vector<prec> &dxdt ,const double t)
    	{
    	    for (int i = 0; i < N; ++i)
    	    {
    	    	prec sum=interaction_sum_chain(i,x);

        		dxdt[2*i]=x[2*i+1];
				dxdt[2*i+1]=sum/I[i]+F[i]*sin(W[i]*t-x[2*i])/I[i]-(G[i]/I[i])*x[2*i+1];      	
        	}
    	}

    	prec interaction_sum_all(int id, const std::vector<prec> &x)
    	{
    		prec sum=0;
    		for (int i = 0; i < N; ++i)
    		{
    			sum=sum+A[id][i]*sin(x[2*i]-x[2*id])/N;
    		}
    		return sum*K;
    	}

    	prec interaction_sum_chain(int id,const std::vector<prec> &x)
    	{
    		prec sum=0;
    	    if(id==0)
    	    {
    	    	sum=sum+A[id][id+1]*sin(x[2*(id+1)]-x[2*id])/N;
    	    }
    	    if(id==N-1)
    	    {
    	    	sum=sum+A[id][id-1]*sin(x[2*(id-1)]-x[2*id])/N;
    	    }
    	    if(id>0 && id<N-1)
    	    {
    	    	sum=sum+A[id][id+1]*sin(x[2*(id+1)]-x[2*id])/N;
    	    	sum=sum+A[id][id-1]*sin(x[2*(id-1)]-x[2*id])/N;
    	    }
    	   
    		return sum*K;
    	}

    	void print_params()
    	{
			std::ofstream txtOut;
			txtOut.open("params.txt");
			txtOut.precision(8);
			txtOut << N << " " << K << std::endl;
			txtOut.close();

    		I.print();
    		F.print();
    		G.print();
    		W.print();
    	}

    	void print_params_to_console()
    	{
    		I.print_to_console();
    		F.print_to_console();
    		G.print_to_console();
    		W.print_to_console();
    	}
};

int main()
{
 	boost::mt19937 rng(static_cast<unsigned int>(std::time(0)));

	std::cout << "Hello World" << std::endl;


	Dinamic P(rng);
	P.print_params();
	P.print_params_to_console();

	return 0;
}