#include "hip/hip_runtime.h"
#include <string>
#include <vector>
#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <cstring>
#include <limits>

#include <sys/stat.h>
#include <boost/numeric/odeint.hpp>
#include <boost/numeric/odeint/external/openmp/openmp.hpp>
#include <boost/random.hpp>

typedef double prec;

#include "Ensemble_param.h"
#include "Ensemble_connections.h"
#include "Ensemble_Dinamic.h"
#include "Ensemble_evolve.h"

void check_stationary(Evolve &e, Dinamic &P)
{
	bool statement=false;
	int count=0;
	while(statement==false)
	{
		e.clean();
		e.run(P,count*10000,(count+1)*10000,100);

		for (int i = 1; i < e.size(); ++i)
		{	
			prec convergence=e.convergence(i);
			if(convergence>0)
			{
				std::cout << i <<": " << convergence << std::endl;
			}
		}

		statement=e.did_converge(49);
		count++;

		e.reset_init();		
	}
	e.print();
	e.print_init();
}

void printU(Evolve &e, Dinamic &P)
{
	std::vector<prec> U(e.size(),0);

	for (int i = 1; i < U.size(); ++i)
	{
		prec aux=e.Amp(i,P);
		if(aux>0)
		{
			U[i]=log(aux);
		}
	}

	std::string name="U";
	mkdir("out", 0777);
	std::ofstream txtOut;
	txtOut.open("out/"+name+".txt");
	txtOut.precision(std::numeric_limits< prec >::max_digits10);

	for (int i = 1; i < U.size()-1; ++i)
	{
		U[i]=(U[i+1]-U[i])/0.0025;
		prec diff=e.Diff(i);
		if(diff>0)
		{
			std::cout <<"U(" << i << ")= " << U[i]<<"; Diff(" << i <<")= "<< diff << "; sin aprox(" << i << ")= " << 100*sin(diff)/diff << std::endl;
			txtOut << i << " " << U[i] << " " << diff << " " << 100*sin(diff)/diff << std::endl;
		}
	}
	txtOut.close();
	
}


int main()
{
 	boost::mt19937 rng(static_cast<unsigned int>(std::time(0)));

 	int N;
 	std::cout << "N: ";
 	std::cin >> N;

    int chunk_size = N/omp_get_max_threads();
    omp_set_schedule( omp_sched_static , chunk_size );

    prec K=16*N;
    prec I=0.1;
    prec G=5;
	Dinamic P(N ,K ,rng ,I ,0 ,1 ,0 ,G ,0 ,1 ,0,"chain"); //N, K, rng, I, sigm_I, F, sigm_F, G, sigm_G, W, sigm_W
	P.print_params();

	
	Evolve e(N, rng);
	//check_stationary(e, P);
	e.run(P,0,50000,100);
	e.print();
	for (int i = 1; i < N; ++i)
	{
		std::cout << i << ": " << e.Amp(i,P) << std::endl;
	}
	//e.print();
	e.print_init();
	printU(e,P);




	return 0;
}