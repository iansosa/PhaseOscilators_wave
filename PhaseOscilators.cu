#include "hip/hip_runtime.h"
#include <string>
#include <vector>
#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <cstring>
#include <limits>

#include <sys/stat.h>
#include <boost/numeric/odeint.hpp>
#include <boost/numeric/odeint/external/openmp/openmp.hpp>
#include <boost/random.hpp>

typedef double prec;

#include "Ensemble_param.h"
#include "Ensemble_connections.h"
#include "Ensemble_Dinamic.h"
#include "Ensemble_evolve.h"


void calc_U(Evolve &e, Dinamic &P, prec dx, std::vector<prec> &U)
{
	for (int i = 1; i < U.size(); ++i)
	{
		prec aux=e.Amp(i,P);
		if(aux>0)
		{
			U[i]=log(aux);
		}
	}

	for (int i = 1; i < U.size()-1; ++i)
	{
		U[i]=(U[i+1]-U[i])/dx;
	}
}

void printU(Evolve &e, Dinamic &P, prec dx)
{
	std::vector<prec> U(e.size(),0);

	calc_U(e,P,dx,U);

	std::string name="U";
	mkdir("out", 0777);
	std::ofstream txtOut;
	txtOut.open("out/"+name+".txt");
	txtOut.precision(std::numeric_limits< prec >::max_digits10);

	for (int i = 1; i < U.size()-1; ++i)
	{
			txtOut << i << " " << U[i] << std::endl;
	}
	txtOut.close();	
}

void print_U_stats(std::vector<std::vector<prec>> &U)
{
	std::vector<prec> U_med(U[0].size(),0);
	std::vector<prec> U_sigm(U[0].size(),0);
	for (int i = 0; i < U[0].size(); ++i) //N
	{
		int norma=0;
		for (int j = 0; j < U.size(); ++j) //promedio
		{
			if(U[j][i]>-1000 && U[j][i]<500)
			{
				norma++;
				U_med[i]=U_med[i]+U[j][i];
			}	
		}
		if(norma>0)
		{
			U_med[i]=U_med[i]/norma;
		}
	}

	for (int i = 0; i < U[0].size(); ++i) //N
	{
		int norma=0;
		for (int j = 0; j < U.size(); ++j) //promedio
		{
			if(U[j][i]>-1000 && U[j][i]<500)
			{
				norma++;
				U_sigm[i]=pow(U[j][i]-U_med[i],2);
			}	
		}
		if(norma>0)
		{
			U_sigm[i]=U_sigm[i]/norma;
		}
		U_sigm[i]=sqrt(U_sigm[i]);
	}
	std::string name="U_stats";
	mkdir("out", 0777);
	std::ofstream txtOut;
	txtOut.open("out/"+name+".txt");
	txtOut.precision(std::numeric_limits< prec >::max_digits10);

	for (int i = 0; i < U_med.size(); ++i)
	{
			txtOut << i << " " << U_med[i] << " " << U_sigm[i] << std::endl;
	}
	txtOut.close();	
}

void promedio_U()
{
 	boost::mt19937 rng(static_cast<unsigned int>(std::time(0)));

 	int N;
 	std::cout << "N: ";
 	std::cin >> N;

  	int prom;
 	std::cout << "Prom: ";
 	std::cin >> prom;

 	std::vector<std::vector<prec> > U(prom);

    int chunk_size = N/omp_get_max_threads();
    omp_set_schedule( omp_sched_static , chunk_size );

    prec conv_factor=5;
    prec dx=0.01;
    dx=dx/conv_factor;

    prec K=conv_factor*conv_factor*N;
    prec I=0.1;
    prec G=5;
	Dinamic P(N ,K ,rng ,I ,I*0.01 ,1 ,0 ,G ,G*0.01 ,1 ,0,"chain"); //N, K, rng, I, sigm_I, F, sigm_F, G, sigm_G, W, sigm_W
	P.init_I_type_rand(true,false);

	Evolve e(N, rng);
	for (int i = 0; i < prom; ++i)
	{
		std::cout << "Iter: (" << i+1 << "/" << prom << ")" << std::endl;
		U[i].resize(N);
		e.run(P,0,50000,100);
		e.calc_convergence();
		calc_U(e,P,dx,U[i]);
		e.clean();
		P.print_params();

		P.generate();
	}
	e.print();

	print_U_stats(U);
}

int main()
{
 	boost::mt19937 rng(static_cast<unsigned int>(std::time(0)));

 	int N;
 	std::cout << "N: ";
 	std::cin >> N;
 	int chunk_size = N/omp_get_max_threads();
 	omp_set_schedule( omp_sched_static , chunk_size );

    prec I=0.1;
    prec G=0.25;
    prec K=N;
	Dinamic P(N ,K ,rng ,I ,I*0 ,1000 ,0 ,G ,G*0 ,1 ,0,"chain"); //N, K, rng, I, sigm_I, F, sigm_F, G, sigm_G, W, sigm_W
	Evolve e(N, rng);

	e.run(P,0,200000,1000);
	e.print();
	e.calc_convergence();
	printU(e,P,0.01);

	return 0;
}