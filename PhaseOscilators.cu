#include "hip/hip_runtime.h"
#include <string>
#include <vector>
#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <cstring>

#include <boost/numeric/odeint.hpp>
#include <boost/random.hpp>

typedef double prec;

class Dinamic 
{
    prec K;
    int N;

    std::vector<prec> &I;
    std::vector<prec> &F;
    std::vector<std::vector<prec>> &A;
    std::vector<prec> &G;
    std::vector<prec> &W;

	public:
    Dinamic(prec in_K ,int in_N ,std::vector<prec> &in_I ,std::vector<std::vector<prec>> &in_A ,std::vector<prec> &in_F ,std::vector<prec> &in_G ,std::vector<prec> &in_W) : K(in_K) , N(in_N) , I(in_I), A(in_A), F(in_F) , G(in_G), W(in_W)
    { }

    void operator() (const std::vector<prec> &x ,std::vector<prec> &dxdt ,const double t)
    {
    	double sum=0;

        for (int i = 0; i < N; ++i)
        {
        	prec sum=interaction_sum_chain(i,x);

        	dxdt[2*i]=x[2*i+1];
			dxdt[2*i+1]=sum/I[i]+F[i]*sin(W[i]*t-x[2*i])/I[i]-(G[i]/I[i])*x[2*i+1];      	
        }
    }

    prec interaction_sum_all(int id, const std::vector<prec> &x)
    {
    	prec sum=0;
    	for (int i = 0; i < N; ++i)
    	{
    		sum=sum+A[id][i]*sin(x[2*i]-x[2*id])/N;
    	}
    	return sum*K;
    }

    prec interaction_sum_chain(int id,const std::vector<prec> &x)
    {
    	prec sum=0;
        if(id==0)
        {
        	sum=sum+A[id][id+1]*sin(x[2*(id+1)]-x[2*id])/N;
        }
        if(id==N-1)
        {
        	sum=sum+A[id][id-1]*sin(x[2*(id-1)]-x[2*id])/N;
        }
        if(id>0 && id<N-1)
        {
        	sum=sum+A[id][id+1]*sin(x[2*(id+1)]-x[2*id])/N;
        	sum=sum+A[id][id-1]*sin(x[2*(id-1)]-x[2*id])/N;
        }
       
    	return sum*K;
    }
};

int main()
{
	std::cout << "Hello World" << std::endl;



	return 0;
}