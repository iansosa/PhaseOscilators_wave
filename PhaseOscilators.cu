#include "hip/hip_runtime.h"
#include <string>
#include <vector>
#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <cstring>

#include <sys/stat.h>
#include <boost/numeric/odeint.hpp>
#include <boost/random.hpp>

typedef double prec;

#include "Ensemble_param.h"
#include "Ensemble_connections.h"

class Dinamic 
{
    prec K;

    Ensemble_param I;
    Ensemble_param F;
    std::vector<std::vector<prec>> A;
    Ensemble_param G;
    Ensemble_param W;

    boost::mt19937 &rng;

	public:
		int N;

    	Dinamic(boost::mt19937 &in_rng) : rng(in_rng), I("I",in_rng), F("F",in_rng), G("G",in_rng), W("W",in_rng)
    	{
			std::ifstream txtIn_check;
			txtIn_check.open("params.txt");
			txtIn_check >> N;
			txtIn_check >> K;
			txtIn_check.close();
    	}

    	Dinamic(int in_N , prec in_K, boost::mt19937 &in_rng, prec mid_I, prec sigma_I, prec mid_F, prec sigma_F, prec mid_G, prec sigma_G, prec mid_W, prec sigma_W) : N(in_N), K(in_K), rng(in_rng), I(in_N,"I",mid_I,sigma_I,in_rng), F(in_N,"F",mid_F,sigma_F,in_rng,false), A(in_N), G(in_N,"G",mid_G,sigma_G,in_rng), W(in_N,"W",mid_W,sigma_W,in_rng,false)
    	{ }

    	void operator() (const std::vector<prec> &x ,std::vector<prec> &dxdt ,const double t)
    	{
    	    for (int i = 0; i < N; ++i)
    	    {
    	    	prec sum=interaction_sum_chain(i,x);

        		dxdt[2*i]=x[2*i+1];
				dxdt[2*i+1]=sum/I[i]+F[i]*sin(W[i]*t-x[2*i])/I[i]-(G[i]/I[i])*x[2*i+1];      	
        	}
    	}

    	prec interaction_sum_all(int id, const std::vector<prec> &x)
    	{
    		prec sum=0;
    		for (int i = 0; i < N; ++i)
    		{
    			sum=sum+A[id][i]*sin(x[2*i]-x[2*id])/N;
    		}
    		return sum*K;
    	}

    	prec interaction_sum_chain(int id,const std::vector<prec> &x)
    	{
    		prec sum=0;
    	    if(id==0)
    	    {
    	    	sum=sum+A[id][id+1]*sin(x[2*(id+1)]-x[2*id])/N;
    	    }
    	    if(id==N-1)
    	    {
    	    	sum=sum+A[id][id-1]*sin(x[2*(id-1)]-x[2*id])/N;
    	    }
    	    if(id>0 && id<N-1)
    	    {
    	    	sum=sum+A[id][id+1]*sin(x[2*(id+1)]-x[2*id])/N;
    	    	sum=sum+A[id][id-1]*sin(x[2*(id-1)]-x[2*id])/N;
    	    }
    	   
    		return sum*K;
    	}

    	void print_params()
    	{
			std::ofstream txtOut;
			txtOut.open("params.txt");
			txtOut.precision(8);
			txtOut << N << " " << K << std::endl;
			txtOut.close();

    		I.print();
    		F.print();
    		G.print();
    		W.print();
    	}

    	void print_params_to_console()
    	{
    		I.print_to_console();
    		F.print_to_console();
    		G.print_to_console();
    		W.print_to_console();
    	}
};

int main()
{
 	boost::mt19937 rng(static_cast<unsigned int>(std::time(0)));

	std::cout << "Hello World" << std::endl;


	//Dinamic P(9 ,1 ,rng ,1 ,0.1 ,2 ,0.2 ,3 ,0.3 ,4 ,0.3);
	//P.print_params();
	Dinamic P(rng);
	P.print_params_to_console();

	Ensemble_connections A(10, "A", rng);
	A.print_to_console();
	A.print();

	return 0;
}